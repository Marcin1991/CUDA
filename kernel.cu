#include "hip/hip_runtime.h"
#include "kernel.h"
#include <iostream>
#include <cmath>

#define FILTER_SIZE 5

/*Kernel blur function -> box blur used defined size matrix*/
__global__ void cudaBlurFilter(int width, int height, unsigned char* in_image, unsigned char* out_image) {

    unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    
    int x = offset % width;
    int y = (offset-x)/width;
    
    int size = FILTER_SIZE;

    if(offset < width*height) {

        float red = 0;
        float green = 0;
        float blue = 0;
        
        int fields_count = 0;
        
        //main loops, count average values of max = size * size pixels
        for(int ox = -size; ox < size+1; ++ox) {
            for(int oy = -size; oy < size+1; ++oy) {
                
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    
                    int currentoffset = (offset+ox+oy*width)*3;
                    
                    out_red += in_image[currentoffset]; 
                    out_green += in_image[currentoffset+1];
                    out_blue += in_image[currentoffset+2];
                    
                    fields_count++;
                }
            }
        }
        
        /*save results to output image array*/
        out_image[offset*3] = red/fields_count;
        out_image[offset*3+1] = green/fields_count;
        out_image[offset*3+2] = blue/fields_count;
        
        }
}


void filter(int width, int height, unsigned char* in_image, unsigned char* out_image) {

    /*malloc two arrays for images on device*/
    unsigned char* dev_input, dev_output;
    hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char));
    hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char));

    /*copy data from host to device memory*/
    hipMemcpy( dev_input, in_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice );
 
    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );

    cudaBlurFilter<<<gridDims, blockDims>>>(width, height, dev_input, dev_output); 

    /*copy results to host*/
    hipMemcpy(out_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost );

    hipFree(dev_input);
    hipFree(dev_output);
}

